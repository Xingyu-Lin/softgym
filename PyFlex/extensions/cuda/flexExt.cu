#include "hip/hip_runtime.h"
// This code contains NVIDIA Confidential Information and is disclosed to you
// under a form of NVIDIA software license agreement provided separately to you.
//
// Notice
// NVIDIA Corporation and its licensors retain all intellectual property and
// proprietary rights in and to this software and related documentation and
// any modifications thereto. Any use, reproduction, disclosure, or
// distribution of this software and related documentation without an express
// license agreement from NVIDIA Corporation is strictly prohibited.
//
// ALL NVIDIA DESIGN SPECIFICATIONS, CODE ARE PROVIDED "AS IS.". NVIDIA MAKES
// NO WARRANTIES, EXPRESSED, IMPLIED, STATUTORY, OR OTHERWISE WITH RESPECT TO
// THE MATERIALS, AND EXPRESSLY DISCLAIMS ALL IMPLIED WARRANTIES OF NONINFRINGEMENT,
// MERCHANTABILITY, AND FITNESS FOR A PARTICULAR PURPOSE.
//
// Information and code furnished is believed to be accurate and reliable.
// However, NVIDIA Corporation assumes no responsibility for the consequences of use of such
// information or for any infringement of patents or other rights of third parties that may
// result from its use. No license is granted by implication or otherwise under any patent
// or patent rights of NVIDIA Corporation. Details are subject to change without notice.
// This code supersedes and replaces all information previously supplied.
// NVIDIA Corporation products are not authorized for use as critical
// components in life support devices or systems without express written approval of
// NVIDIA Corporation.
//
// Copyright (c) 20132017 NVIDIA Corporation. All rights reserved.

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <vector>
#include <limits>
#include <algorithm>

#include "../../core/core.h"
#include "../../core/maths.h"

#include "../../include/NvFlex.h"
#include "../../include/NvFlexExt.h"

#define CudaCheck(x) { hipError_t err = x; if (err != hipSuccess) { printf("Cuda error: %d in %s at %s:%d\n", err, #x, __FILE__, __LINE__); assert(0); } }

static const int kNumThreadsPerBlock = 256;

struct NvFlexExtForceFieldCallback
{
	NvFlexExtForceFieldCallback(NvFlexSolver* solver) : mSolver(solver)
	{
		// force fields
		mForceFieldsCpu = NULL;
		mForceFieldsGpu = NULL;
		mMaxForceFields = 0;
		mNumForceFields = 0;

	}

	~NvFlexExtForceFieldCallback()
	{
		// force fields
		CudaCheck(hipHostFree(mForceFieldsCpu));
		CudaCheck(hipFree(mForceFieldsGpu));
	}
	
	NvFlexExtForceField* mForceFieldsCpu;	// pinned host copy for async transfer
	NvFlexExtForceField* mForceFieldsGpu; // device copy

	int mMaxForceFields;
	int mNumForceFields;

	NvFlexSolver* mSolver;
};

NvFlexExtForceFieldCallback* NvFlexExtCreateForceFieldCallback(NvFlexSolver* solver)
{
	return new NvFlexExtForceFieldCallback(solver);	
}

void NvFlexExtDestroyForceFieldCallback(NvFlexExtForceFieldCallback* callback)
{
	delete callback;
}


__global__ void UpdateForceFields(int numParticles, const Vec4* __restrict__ positions, Vec4* __restrict__ velocities, const NvFlexExtForceField* __restrict__ forceFields, int numForceFields, float dt)
{
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	
	for (int f = 0; f < numForceFields; f++)
	{
		const NvFlexExtForceField& forceField = forceFields[f];

		if (i < numParticles)
		{
			const int index = i;

			Vec4 p = positions[index];
			Vec3 v = Vec3(velocities[index]);

			Vec3 localPos = Vec3(p.x, p.y, p.z) - Vec3(forceField.mPosition[0], forceField.mPosition[1], forceField.mPosition[2]);

			float length = Length(localPos);
			if (length >= forceField.mRadius)
			{
				continue;
			}
			
			Vec3 fieldDir;
			if (length > 0.0f)
			{
				fieldDir = localPos / length;
			}
			else
			{
				fieldDir = localPos;
			}

			// If using linear falloff, scale with distance.
			float fieldStrength = forceField.mStrength;
			if (forceField.mLinearFalloff)
			{
				fieldStrength *= (1.0f - (length / forceField.mRadius));
			}

			// Apply force
			Vec3 force = localPos * fieldStrength;

			float unitMultiplier;
			if (forceField.mMode == eNvFlexExtModeForce)
			{
				unitMultiplier = dt * p.w; // time/mass
			} 
			else if (forceField.mMode == eNvFlexExtModeImpulse)
			{
				unitMultiplier = p.w; // 1/mass
			}
			else if (forceField.mMode == eNvFlexExtModeVelocityChange)
			{
				unitMultiplier = 1.0f;
			}

			Vec3 deltaVelocity = fieldDir * fieldStrength * unitMultiplier;
			velocities[index] = Vec4(v + deltaVelocity, 0.0f);
		}
	}
}

void ApplyForceFieldsCallback(NvFlexSolverCallbackParams params)
{
	// callbacks always have the correct CUDA device set so we can safely launch kernels without acquiring

	NvFlexExtForceFieldCallback* c = (NvFlexExtForceFieldCallback*)params.userData;

	if (params.numActive && c->mNumForceFields)
	{
		const int kNumBlocks = (params.numActive+kNumThreadsPerBlock-1)/kNumThreadsPerBlock;

		UpdateForceFields<<<kNumBlocks, kNumThreadsPerBlock>>>(
			params.numActive,
	   		(Vec4*)params.particles,
	   		(Vec4*)params.velocities,
			c->mForceFieldsGpu,
			c->mNumForceFields,
			params.dt);
	}
}

void NvFlexExtSetForceFields(NvFlexExtForceFieldCallback* c, const NvFlexExtForceField* forceFields, int numForceFields)
{
	// re-alloc if necessary
	if (numForceFields > c->mMaxForceFields)
	{
		CudaCheck(hipHostFree(c->mForceFieldsCpu));
		CudaCheck(hipHostMalloc(&c->mForceFieldsCpu, sizeof(NvFlexExtForceField)*numForceFields));

		CudaCheck(hipFree(c->mForceFieldsGpu));
		CudaCheck(hipMalloc(&c->mForceFieldsGpu, sizeof(NvFlexExtForceField)*numForceFields));


		c->mMaxForceFields = numForceFields;
	}
	c->mNumForceFields = numForceFields;

	if (numForceFields > 0)
	{
		// copy to pinned host memory
		memcpy(c->mForceFieldsCpu, forceFields, numForceFields*sizeof(NvFlexExtForceField));

		hipMemcpyKind kind = hipMemcpyHostToDevice;
		CudaCheck(hipMemcpyAsync(c->mForceFieldsGpu, &c->mForceFieldsCpu[0], numForceFields*sizeof(NvFlexExtForceField), kind, 0));
	}

	NvFlexSolverCallback callback;
	callback.function = ApplyForceFieldsCallback;
	callback.userData = c;
	
	// register a callback to calculate the forces at the end of the time-step
	NvFlexRegisterSolverCallback(c->mSolver, callback, eNvFlexStageUpdateEnd);
}
